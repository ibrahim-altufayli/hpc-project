
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <hip/hip_complex.h>
#include <chrono>
#include <string.h>
#include <cmath>

// Ranges of the set
#define MIN_X -2
#define MAX_X 1
#define MIN_Y -1
#define MAX_Y 1



#define DEGREE 2        // Degree of the polynomial

using namespace std;

__global__ void calcMandelbrotPxl(int* image, int width, int height, int step, int min_x, int min_y, int iterations, int dutySize){
    //I should determine which portion of positions I should deal with
    int startPos = threadIdx.x + blockIdx.x * blockDim.x;
    int endPos = startPos + dutySize;
    for (int pos = startPos; pos < endPos; pos++)
    {

        if (pos < width * height){

        image[pos] = pos;

        /*int row = pos / width;
        int col = pos % width;
        cuDoubleComplex c = make_cuDoubleComplex(col * step + min_x, row * step + min_y);

        // z = z^2 + c
        cuDoubleComplex z= make_cuDoubleComplex(0, 0);
        for (int i = 1; i <= iterations; i++)
        {
            z = cuCadd(cuCmul(z, z), c);

            // If it is convergent
            if (cuCreal(z)*cuCreal(z) + cuCimag(z)*cuCimag(z) >= 4)
            {
                image[pos] = i;
                break;
            }
        }*/
        }

    }
}

double calc_rmse(int * imageGen, string refImagePath){
    ifstream file(refImagePath.c_str()); 
    int row = 0;
    int width = 0;
    string line;
    string delimiter = ",";
    int sumOfDiff = 0;
    while (std::getline(file, line))
    {
        size_t delimiterPos = 0;
        int pxlCol = 0;
        while ((delimiterPos = line.find(delimiter)) != string::npos) {
            string pxlValueStr = line.substr(0, delimiterPos);
            char* p;
            int pxlValue = strtol(pxlValueStr.c_str(), &p, 10);
            line.erase(0, delimiterPos + delimiter.length());
            int pxl1DPos = row * width + pxlCol;
            sumOfDiff += pow(pxlValue - imageGen[pxl1DPos], 2);
            pxlCol +=1;
        }
        //handel last pixel
        string pxlValueStr = line;
        char* p;
        int pxlValue = strtol(pxlValueStr.c_str(), &p, 10);
        int pxl1DPos = row * width + pxlCol;
        sumOfDiff += pow(pxlValue - imageGen[pxl1DPos], 2);
        pxlCol +=1;
        if(width == 0){
            width = pxlCol;
        }
        row += 1;
    }
    int height = row;
    file.close();
    cout<<"width: "<<width<<" height: "<<height<<endl;
    return sqrt(sumOfDiff/(height * width));
}

int main(int argc, char **argv)
{
    int ITERATIONS = 1000, RESOLUTION = 1000;
    // Image ratio
    int RATIO_X = MAX_X - MIN_X;
    int RATIO_Y = MAX_Y - MIN_Y;


    if(argc > 1){
        char *p;
        RESOLUTION = strtol(argv[1], &p, 10);
        if (*p != '\0'){
            cout << "Please use only integer values for RESOLUTION" << endl;
            return -1;
        }
        if(argc > 2){
            p = NULL;
            ITERATIONS = strtol(argv[2], &p, 10);
            if(*p != '\0'){
                cout << "Please use only integer values for ITERATIONS" << endl;
                return -1;
            }

        }
    }
    cout<< "Image Resolution: "<< RESOLUTION<<endl;
    cout<< "#Iterations: "<< ITERATIONS <<endl;
    

    int WIDTH = RATIO_X * RESOLUTION;
    int HEIGHT = RATIO_Y * RESOLUTION;

    double STEP = ((double)RATIO_X / WIDTH);

    
    int N = WIDTH * HEIGHT;
    int * image = (int*)malloc( N * sizeof(int) );
    int *dev_image;

    const auto start = chrono::steady_clock::now();
    dim3 threads(32);
    dim3 blocks ( (N+threads.x-1)/threads.x );
    int dutySize = N/(threads.x * blocks.x);

    hipMalloc( (void**)&dev_image, N * sizeof(int) );
    hipMemcpy(dev_image, image, N * sizeof(int), hipMemcpyHostToDevice);

    calcMandelbrotPxl<<<blocks,threads>>>( dev_image, WIDTH, HEIGHT, STEP, MIN_X, MIN_Y, ITERATIONS, dutySize);
    

    hipDeviceSynchronize();
    hipMemcpy(image, dev_image, N * sizeof(int), hipMemcpyDeviceToHost);




      const auto end = chrono::steady_clock::now();
    cout << "Time elapsed: "
         << chrono::duration_cast<chrono::milliseconds>(end - start).count()
         << " milliseconds." << endl;

    
    
        ofstream results_out;
        results_out.open("builds/time_results.txt", ios::app);
        if(!results_out.is_open()){
            results_out.open("builds/time_results.txt", ios::trunc);
        }

	       double rmse = calc_rmse(image, "imgs/img_ref_" + to_string(RESOLUTION));
		  cout<<"RMSE: "<<rmse<<endl;

        results_out<<chrono::duration_cast<chrono::milliseconds>(end - start).count()
         << ","<< RESOLUTION<<","<<ITERATIONS<< ','<<rmse<<endl;

        results_out.close();
    

    // Write the result to a file
    ofstream matrix_out;

    
    string imgFilePath = "imgs/img_";
    imgFilePath.append(to_string(RESOLUTION)+"_"+to_string(ITERATIONS));

    cout<<imgFilePath<<endl;
    matrix_out.open(imgFilePath.c_str(), ios::trunc);
    if (!matrix_out.is_open())
    {
        cout << "Unable to open file." << endl;
        return -2;
    }

    for (int row = 0; row < HEIGHT; row++)
    {
        for (int col = 0; col < WIDTH; col++)
        {
            matrix_out << image[row * WIDTH + col];

            if (col < WIDTH - 1)
                matrix_out << ',';
        }
        if (row < HEIGHT - 1)
            matrix_out << endl;
    }
    matrix_out.close();

    delete[] image; // It's here for coding style, but useless
    return 0;
}

